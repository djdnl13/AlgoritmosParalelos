
#include <hip/hip_runtime.h>
#include <iostream>

typedef int t_number;

using namespace std;

__global__ void Matrix_Addition(t_number * dev_a , t_number * dev_b , t_number * dev_c, size_t n)
{
     size_t tid = blockIdx.x*blockDim.x + threadIdx.x;

     if (tid < n*n)
         dev_c[tid] = dev_a[tid] + dev_b[tid];
}

__global__ void Matrix_Addition_Row(t_number * dev_a , t_number * dev_b , t_number * dev_c, size_t n)
{
    size_t tid = blockIdx.x*blockDim.x + threadIdx.x;
    for(size_t i=0 ; i<n ; ++i)
         dev_c[tid+i] = dev_a[tid+i] + dev_b[tid+i];
}

__global__ void Matrix_Addition_Column(t_number * dev_a , t_number * dev_b , t_number * dev_c, size_t n)
{
    size_t tid = blockIdx.x*blockDim.x + threadIdx.x;
    for(size_t i=0 ; i<n ; ++i)
         dev_c[tid+i*n] = dev_a[tid+i*n] + dev_b[tid+i*n];
}

int main()
{
      t_number * Host_a, * Host_b, * Host_c;
      t_number * dev_a , * dev_b, * dev_c ;
      size_t n = 10;
      t_number size = n*n;

      hipMalloc((void **)&dev_a , size*sizeof(t_number));
      hipMalloc((void **)&dev_b , size*sizeof(t_number));
      hipMalloc((void **)&dev_c , size*sizeof(t_number));

      Host_a = new t_number[size];
      Host_b = new t_number[size];
      Host_c = new t_number[size];

      for (size_t i = 0; i<size ; ++i)
      {
            Host_a[i] = i ;
            Host_b[i] = i*2 ;
      }

      hipMemcpy(dev_a , Host_a , size*sizeof(t_number) , hipMemcpyHostToDevice);
      hipMemcpy(dev_b , Host_b , size*sizeof(t_number) , hipMemcpyHostToDevice);

//      Matrix_Addition<<<N, N>>>(dev_a , dev_b , dev_c, n);
//      Matrix_Addition_Row<<<n, 1>>>(dev_a , dev_b , dev_c, n);
      Matrix_Addition_Column<<<1, n>>>(dev_a , dev_b , dev_c, n);

      hipMemcpy(Host_c , dev_c , size*sizeof(t_number) , hipMemcpyDeviceToHost);

      for(size_t i=0 ; i<size ; ++i)
      {
          if(i%n == 0)
             cout << endl;
          cout << Host_c[i] << " " ;          
      }
      cout << endl;

      hipFree(dev_a) ;
      hipFree(dev_b) ;
      hipFree(dev_c) ;

      return 0 ;
}
