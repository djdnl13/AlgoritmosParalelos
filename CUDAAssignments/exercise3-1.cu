
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

	// B
__global__ void kerner_matrix_addition(float** A, float** B, float** C, int n)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	if(i<n and j<n)
		A[i][j] = B[i][j] + C[i][j];
}

	// C
__global__ void kerner_matrix_addition_row(float** A, float** B, float** C, int n)
{
	int i = threadIdx.x;

	for(int j=0 ; j<n ; ++j)
		A[i][j] = B[i][j] + C[i][j];
}

	// D
__global__ void kerner_matrix_addition_column(float** A, float** B, float** C, int n)
{
	int j = threadIdx.y;

	for(int i=0 ; i<n ; ++i)
		A[i][j] = B[i][j] + C[i][j];
}

void vecAdd(float ** A, float ** B, float ** C, int n)
{
	int size = n*n*sizeof(float);
	float **d_A, **d_B, **d_C;		

	hipMalloc((void **) &d_B, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_C, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_A, size);

	//vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);
	//dim3 threadsPerBlock(n, n);

	// Kernel invocation code
	kerner_matrix_addition<<<(1,1), (n,n)>>>(d_A, d_B, d_C, n);
	
	/*kerner_matrix_addition_row<<<1, (n,1)>>>(d_A, d_B, d_C, n);
	kerner_matrix_addition_column<<<1, (1,n)>>>(d_A, d_B, d_C, n);
	*/

	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);

 		// Free device memory for A, B, C
	hipFree(d_A); hipFree(d_B); hipFree (d_C);
}

int main()
{
	float ** A, ** B, ** C;
	int n = 256;
	
	A = new float * [n];
	B = new float * [n];
	C = new float * [n];

	for(size_t i=0 ; i<n ; i++)
	{
		A[i] = new float[n];
		B[i] = new float[n];
		C[i] = new float[n];

		for(size_t j=0 ; j<n ; j++)
		{
			A[i][j] = 1;
			B[i][j] = 2;
			C[i][j] = 0;
		}
	}

	vecAdd(A, B, C, n);

	for(size_t i=0 ; i<n ; i++)
	{
		for(size_t j=0 ; j<n ; j++)
		{
			cout << A[i][j] << " ";
		}
		cout << endl;		
	}



	return 0;
}