#include "hip/hip_runtime.h"
	__global__ void kerner_matrix_addition(float * A, float ** B, float* C, int n)
	{
		int i = threadIdx.x;

		if(i>=n)
			return;

		A[i] = 0;
		for(size_t j=0; j<n ; ++j)
			A[i] += B[i][j] * C[i];
	}

	void vecAdd(float * A, float ** B, float * C, int n)
	{
		int size_vector = n*sizeof(float);
		int size_matrix = n*n*sizeof(float);
		float **d_A, **d_B, **d_C;

		hipMalloc((void **) &d_B, size_matrix);
		hipMemcpy(d_A, A, size_matrix, hipMemcpyHostToDevice);

		hipMalloc((void **) &B_C, size_vector);
		hipMemcpy(d_B, B, size_vector, hipMemcpyHostToDevice);

		hipMalloc((void **) &d_A, size_vector);
		
		//vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);		
		kerner_matrix_addition<<<1, n>>>(d_A, d_B, d_C, n, m);		

		hipMemcpy(A, d_A, size_vector, hipMemcpyDeviceToHost);

 		// Free device memory for A, B, C
		hipFree(d_A); hipFree(d_B); hipFree (d_C);
	}