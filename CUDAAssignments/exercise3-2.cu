
#include <hip/hip_runtime.h>
#include <iostream>

typedef int t_number;

using namespace std;

__global__ void Matrix_Vector_Multiplication(t_number * dev_a , t_number * dev_b , t_number * dev_c, size_t n)
{
	size_t tid = blockIdx.x*blockDim.x + threadIdx.x;

	if(tid < n)
	{
		dev_a[tid] = 0;
		for(size_t i=0 ; i<n ; ++i)
			dev_a[tid] += dev_b[n*i+tid] * dev_c[i];
	}
}

int main()
{
	size_t n = 10;
	size_t size_vector = n;
	size_t size_matrix = n*n;
	t_number * host_a, * host_b, * host_c;
	t_number * dev_a, * dev_b, * dev_c;

	
	host_a = new t_number[size_vector];
	host_b = new t_number[size_matrix];
	host_c = new t_number[size_vector];
	
	for(size_t i=0; i<size_vector ; ++i)
		host_c[i] = 1 ;

	for(size_t i=0 ; i<size_matrix ; ++i)
		host_b[i] = 1;

	hipMalloc((void **)&dev_a , size_vector*sizeof(t_number));
	hipMalloc((void **)&dev_b , size_matrix*sizeof(t_number));
	hipMalloc((void **)&dev_c , size_vector*sizeof(t_number));

	hipMemcpy(dev_b, host_b, size_matrix*sizeof(t_number), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, host_c, size_vector*sizeof(t_number), hipMemcpyHostToDevice);
		
	Matrix_Vector_Multiplication<<<1, n>>>(dev_a, dev_b, dev_c, n);

	hipMemcpy(host_a, dev_a, size_vector*sizeof(t_number), hipMemcpyDeviceToHost);

	for(size_t i=0; i<size_vector ; ++i)
		cout << host_a[i] << " ";
	cout << endl;
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
