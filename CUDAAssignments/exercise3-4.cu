#include "hip/hip_runtime.h"
/*
Part 1 of the function allocates space in the device (GPU) memory to hold
copies of the A, B, and C vectors, and copies the vectors from the host
memory to the device memory.
*/
	int size = n*sizeof(float);
	float *d_A, *d_B, *d_C;

	hipMalloc((void **) &d_B, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_C, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_A, size);
/*
Part 2 launches parallel execution of the
actual vector addition kernel on the device. 
*/
	vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);